#include "hip/hip_runtime.h"
// includes, system
#include <stdio.h>
#include <assert.h>

__global__ void myFirstKernel( int* d_a )
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	d_a[index] = blockIdx.x * 1000 + threadIdx.x;
}

extern "C" void checkCUDAError(const char *msg)
{
    hipError_t err = hipGetLastError();
    if( hipSuccess != err) 
    {
        fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString( err) );
        exit(-1);
    }                         
}

extern "C" void freeParticleMemory( int *h_a, int *d_a )
{
    // free device memory
    hipFree( d_a );

    // free host memory
    hipHostFree( h_a );
}


extern "C" void finishComputation( int numBlocks, int numThreadsPerBlock, int *h_a, int *d_a )
{
    size_t memSize = numBlocks * numThreadsPerBlock * sizeof(int);

    // block until the device has completed
    hipDeviceSynchronize();

    // check if kernel execution generated an error
    checkCUDAError("kernel execution");

    // Part 4 of 5: device to host copy
    hipMemcpy( h_a, d_a, memSize, hipMemcpyDeviceToHost );

    // Check for any CUDA errors
    checkCUDAError("hipMemcpy");
}

extern "C" void allocateParticleMemory( int numBlocks, int numThreadsPerBlock, void **h_a, void **d_a )
{
    size_t memSize = numBlocks * numThreadsPerBlock * sizeof(int);
    hipHostMalloc( h_a, memSize);
    hipMalloc( d_a, memSize );
}

extern "C" void callMyFirstKernel( int numBlocks, int numThreadsPerBlock, int* d_a )
{
    // configure and launch kernel
    dim3 dimGrid( numBlocks );
    dim3 dimBlock( numThreadsPerBlock );
    myFirstKernel<<< dimGrid , dimBlock >>>( d_a );
}
