#include "hip/hip_runtime.h"

#include <assert.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>
#include "obs_math.h"
#include "filter_math.h"
#include "filter_io.h"
#include "convert.h"

#define NUM_PARTICLES 100000
#define MAX_RANGE 20000 // meters
#define MAX_VEL 15 // meters per second

#define MAX_POS_PERTURB 1000.0 // meters
#define MAX_VEL_PERTURB    2.0 // meters per second

#define MEAN_MANEUVER_TIME 3600 // seconds

#define INITIAL_MAX_WAYPOINTS 10

#define OUTPUT_NAME "particles.out"

// particles in host memory
float *h_x_pos; // meters
float *h_y_pos; // meters
float *h_x_vel; // meters/second
float *h_y_vel; // meters/second
float *h_weight;

// particles in device memory
float *d_x_pos; // meters
float *d_y_pos; // meters
float *d_x_vel; // meters/second
float *d_y_vel; // meters/second
float *d_weight;

void h_init_particle_mem( int );
void h_free_particle_mem( );

void d_init_particle_mem( int );
void d_free_particle_mem( );

void copy_particles_device_to_host( int );

void print_particles( int, int );
void print_particle( int );

void checkCUDAError( const char* );

// CUDA kernel function : initialize a particle
__global__ void init_particle_val( )
{
  int index = blockIdx.x * blockDim.x + threadIdx.x;

  d_x_pos[index]  = frand( -MAX_RANGE, MAX_RANGE );
  d_y_pos[index]  = frand( -MAX_RANGE, MAX_RANGE );
  d_x_vel[index]  = frand( -MAX_VEL, MAX_VEL );
  d_y_vel[index]  = frand( -MAX_VEL, MAX_VEL );
  d_weight[index] = 1.0;
}


int main( int argc, char* argv )
{
  srand( time( NULL ) );

  struct waypoint_list *waypoints1 = read_waypoints( "data/waypoints1.txt" );
  struct waypoint_list *waypoints2 = read_waypoints( "data/waypoints2.txt" );

  printf("Sensor Waypoints:\n");
  print_waypoints( waypoints1 );

  printf("Target Waypoints:\n");
  print_waypoints( waypoints2 );

  struct observation_list *range_obs_list = generate_observations( waypoints1, waypoints2, 2, 100, 0.0, 500.0, 2000.0 );
  struct observation_list *azimuth_obs_list = generate_observations( waypoints1, waypoints2, 1, fromDegrees(8.0), 0.0, 100.0, 2000.0 );
  struct observation_list *obs_list = combine_observations( range_obs_list, azimuth_obs_list );
  printf("Observations:\n");
  print_observations( obs_list );

  h_init_particle_mem( NUM_PARTICLES );

  // define grid and block size
  int numThreadsPerBlock = 256;

  // compute number of blocks needed based on array size and desired block size
  int numBlocks = NUM_PARTICLES / numThreadsPerBlock;

  // launch kernel
  dim3 dimGrid(numBlocks);
  dim3 dimBlock(numThreadsPerBlock);
  init_particle_val<<< dimGrid, dimBlock >>>( );

  // block until the device has completed kernel execution
  hipDeviceSynchronize();

  // check if the init_particle_val kernel generated errors
  checkCUDAError("init_particle_val");

  print_particles( NUM_PARTICLES , 1 );

  copy_particles_device_to_host( NUM_PARTICLES );

  print_particles( NUM_PARTICLES , 1 );
}

void copy_particles_device_to_host( int num )
{
  int size = sizeof( float ) * num;

  hipMemcpy( h_x_pos, d_x_pos, size, hipMemcpyDeviceToHost );
  hipMemcpy( h_y_pos, d_y_pos, size, hipMemcpyDeviceToHost );
  hipMemcpy( h_x_vel, d_x_vel, size, hipMemcpyDeviceToHost );
  hipMemcpy( h_y_vel, d_y_vel, size, hipMemcpyDeviceToHost );
  hipMemcpy( h_weight, d_weight, size, hipMemcpyDeviceToHost );
}

// allocate memory for num particles on host
void h_init_particle_mem( int num )
{
  int size = sizeof( float ) * num;

  h_x_pos  = ( float* ) malloc( size );
  h_y_pos  = ( float* ) malloc( size );
  h_x_vel  = ( float* ) malloc( size );
  h_y_vel  = ( float* ) malloc( size );
  h_weight = ( float* ) malloc( size );
}

// allocate memory for num particles on device
void d_init_particle_mem( int num )
{
  int size = sizeof( float ) * num;

  hipMalloc( (void **) &d_x_pos, size );
  hipMalloc( (void **) &d_y_pos, size );
  hipMalloc( (void **) &d_x_vel, size );
  hipMalloc( (void **) &d_y_vel, size );
  hipMalloc( (void **) &d_weight, size );
}

// free particle memory on host
void h_free_particle_mem( )
{
  free( h_x_pos );
  free( h_y_pos );
  free( h_x_vel );
  free( h_y_vel );
  free( h_weight );
}

// free particle memory on device
void d_free_particle_mem( )
{
  hipFree( d_x_pos );
  hipFree( d_y_pos );
  hipFree( d_x_vel );
  hipFree( d_y_vel );
  hipFree( d_weight );
}

// function adapted from CUDA Technical Training
void checkCUDAError(const char *msg)
{
    hipError_t err = hipGetLastError();
    if( hipSuccess != err) 
    {
        fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString( err) );
        exit(EXIT_FAILURE);
    }                         
}

// writes the current set of particles out to disk as
// a tab delimited text file
void write_particles( char* out_name, int num, int downsample )
{
  FILE* file = fopen( out_name, "w" );

  int i;

  for ( i = 0 ; i < num ; i = i + downsample )
  {
    fprintf( file, "%f\t%f\t%f\t%f\t%f\n", h_x_pos[i], h_y_pos[i], h_x_vel[i], h_y_vel[i], h_weight[i] );
  }

  int success = fclose( file );
}

// prints information on all particles to the console
void print_particles( int num, int downsample )
{
  int i;

  for ( i = 0 ; i < num ; i = i + downsample )
  {
    print_particle( i );
  }
}
