#include "hip/hip_runtime.h"

#include <assert.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>
#include "obs_math.h"
#include "device_filter_math.h"
#include "filter_io.h"
#include "convert.h"

#define NUM_PARTICLES 100000
#define MAX_RANGE 20000 // meters
#define MAX_VEL 15 // meters per second

#define MAX_POS_PERTURB 1000.0 // meters
#define MAX_VEL_PERTURB    2.0 // meters per second

#define MEAN_MANEUVER_TIME 3600 // seconds

#define INITIAL_MAX_WAYPOINTS 10

#define OUTPUT_NAME "particles.out"

// particles in host memory
float *h_x_pos; // meters
float *h_y_pos; // meters
float *h_x_vel; // meters/second
float *h_y_vel; // meters/second
float *h_weight;
float *h_seed; // random seed for particle

// particles in device memory
float *d_x_pos; // meters
float *d_y_pos; // meters
float *d_x_vel; // meters/second
float *d_y_vel; // meters/second
float *d_weight;
float *d_seed; // random seed for particle

// functions from filter_math.h
__device__ float d_frand0( float max );
__device__ float d_frand( float min, float max );

void h_init_particle_mem( int );
void h_free_particle_mem( );
void h_init_seed( int num );

void d_init_particle_mem( int );
void d_free_particle_mem( );

void copy_particles_device_to_host( int );
void copy_particles_host_to_device( int );

void print_particles( int, int );
void print_particle( int );

void checkCUDAError( const char* );

// CUDA kernel function : initialize a particle
__global__ void init_particle_val( float *d_x_pos, float *d_y_pos, float *d_x_vel, float *d_y_vel, float *d_weight, float *d_seed )
{
  int index = blockIdx.x * blockDim.x + threadIdx.x;

  int seed = d_seed[index];

  d_x_pos[index]  = device_frand( seed, -MAX_RANGE, MAX_RANGE );
  seed = device_lcg_rand( seed );
  d_y_pos[index]  = device_frand( seed, -MAX_RANGE, MAX_RANGE );
  seed = device_lcg_rand( seed );
  d_x_vel[index]  = device_frand( seed, -MAX_VEL, MAX_VEL );
  seed = device_lcg_rand( seed );
  d_y_vel[index]  = device_frand( seed, -MAX_VEL, MAX_VEL );
  seed = device_lcg_rand( seed );
  d_weight[index] = 1.0;

  d_seed[index] = seed;
}


int main( int argc, char** argv )
{
  srand( time( NULL ) );

  struct waypoint_list *waypoints1 = read_waypoints( "data/waypoints1.txt" );
  struct waypoint_list *waypoints2 = read_waypoints( "data/waypoints2.txt" );

  printf("Sensor Waypoints:\n");
  print_waypoints( waypoints1 );

  printf("Target Waypoints:\n");
  print_waypoints( waypoints2 );

  struct observation_list *range_obs_list = generate_observations( waypoints1, waypoints2, 2, 100, 0.0, 500.0, 2000.0 );
  struct observation_list *azimuth_obs_list = generate_observations( waypoints1, waypoints2, 1, fromDegrees(8.0), 0.0, 100.0, 2000.0 );
  struct observation_list *obs_list = combine_observations( range_obs_list, azimuth_obs_list );
  printf("Observations:\n");
  print_observations( obs_list );

  h_init_particle_mem( NUM_PARTICLES );

  // we could init the particle seeds from a function of their thread index
  // but we should avoid that kind of linear dependence between the seeds
  // so initialize the seeds on the host and copy them to the device
  h_init_seed( NUM_PARTICLES );

  d_init_particle_mem( NUM_PARTICLES );

  // copy the seeds generated on the host to the device
  copy_particles_host_to_device( NUM_PARTICLES );

  // define grid and block size
  int numThreadsPerBlock = 256;

  // compute number of blocks needed based on array size and desired block size
  int numBlocks = NUM_PARTICLES / numThreadsPerBlock;

  // launch kernel
  dim3 dimGrid(numBlocks);
  dim3 dimBlock(numThreadsPerBlock);
  init_particle_val<<< dimGrid, dimBlock >>>( d_x_pos, d_y_pos, d_x_vel, d_y_vel, d_weight, d_seed );

  // block until the device has completed kernel execution
  hipDeviceSynchronize();

  // check if the init_particle_val kernel generated errors
  checkCUDAError("init_particle_val");

  print_particles( NUM_PARTICLES , 1 );

  copy_particles_device_to_host( NUM_PARTICLES );

  print_particles( NUM_PARTICLES , 1 );
}

void h_init_seed( int num )
{
  int i;

  for ( i = 0 ; i < num ; i++ )
  {
    h_seed[i] = rand();
  }
}

void copy_particles_host_to_device( int num )
{
  int size = sizeof( float ) * num;

  hipMemcpy( h_x_pos, d_x_pos, size, hipMemcpyHostToDevice );
  hipMemcpy( h_y_pos, d_y_pos, size, hipMemcpyHostToDevice );
  hipMemcpy( h_x_vel, d_x_vel, size, hipMemcpyHostToDevice );
  hipMemcpy( h_y_vel, d_y_vel, size, hipMemcpyHostToDevice );
  hipMemcpy( h_weight, d_weight, size, hipMemcpyHostToDevice );
  hipMemcpy( h_seed, d_seed, size, hipMemcpyHostToDevice );
}

void copy_particles_device_to_host( int num )
{
  int size = sizeof( float ) * num;

  hipMemcpy( h_x_pos, d_x_pos, size, hipMemcpyDeviceToHost );
  hipMemcpy( h_y_pos, d_y_pos, size, hipMemcpyDeviceToHost );
  hipMemcpy( h_x_vel, d_x_vel, size, hipMemcpyDeviceToHost );
  hipMemcpy( h_y_vel, d_y_vel, size, hipMemcpyDeviceToHost );
  hipMemcpy( h_weight, d_weight, size, hipMemcpyDeviceToHost );
  hipMemcpy( h_seed, d_seed, size, hipMemcpyDeviceToHost );
}

// allocate memory for num particles on host
void h_init_particle_mem( int num )
{
  int size = sizeof( float ) * num;

  h_x_pos  = ( float* ) malloc( size );
  h_y_pos  = ( float* ) malloc( size );
  h_x_vel  = ( float* ) malloc( size );
  h_y_vel  = ( float* ) malloc( size );
  h_weight = ( float* ) malloc( size );
  h_seed = ( float* ) malloc( size );
}

// allocate memory for num particles on device
void d_init_particle_mem( int num )
{
  int size = sizeof( float ) * num;

  hipMalloc( (void **) &d_x_pos, size );
  hipMalloc( (void **) &d_y_pos, size );
  hipMalloc( (void **) &d_x_vel, size );
  hipMalloc( (void **) &d_y_vel, size );
  hipMalloc( (void **) &d_weight, size );
  hipMalloc( (void **) &d_seed, size );
}

// free particle memory on host
void h_free_particle_mem( )
{
  free( h_x_pos );
  free( h_y_pos );
  free( h_x_vel );
  free( h_y_vel );
  free( h_weight );
  free( h_seed );
}

// free particle memory on device
void d_free_particle_mem( )
{
  hipFree( d_x_pos );
  hipFree( d_y_pos );
  hipFree( d_x_vel );
  hipFree( d_y_vel );
  hipFree( d_weight );
  hipFree( d_seed );
}

// function adapted from CUDA Technical Training
void checkCUDAError(const char *msg)
{
    hipError_t err = hipGetLastError();
    if( hipSuccess != err) 
    {
        fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString( err) );
        exit(EXIT_FAILURE);
    }                         
}

// writes the current set of particles out to disk as
// a tab delimited text file
void write_particles( char* out_name, int num, int downsample )
{
  FILE* file = fopen( out_name, "w" );

  int i;

  for ( i = 0 ; i < num ; i = i + downsample )
  {
    fprintf( file, "%f\t%f\t%f\t%f\t%f\n", h_x_pos[i], h_y_pos[i], h_x_vel[i], h_y_vel[i], h_weight[i] );
  }

  int success = fclose( file );
}

// prints information on all particles to the console
void print_particles( int num, int downsample )
{
  int i;

  for ( i = 0 ; i < num ; i = i + downsample )
  {
    print_particle( i );
  }
}









// implementation based on examples from:
// http://en.wikipedia.org/wiki/Linear_congruential_generator
// java.util.Random.next()
__device__ int device_lcg_rand( int x )
{
  // bitwise and with mask is equivalent to mod 2^mask_bits
  long xl = (x * a + c) & mask;
  // we have generated mask_bits but only need result_bits
  // use the highest order bits because they have longer periods
  return (int) ( xl >> ( mask_bits - result_bits ) );
}

// Return a random float value evenly distributed between 0 and max
// Because cuda threads will keep track of their own seeds, this function
// and the others like it are not random. They simply take an already
// generated random value (seed) and transform it in some way.
// Seed values should be generated by repeated calls to device_lcg_rand().
__device__ float device_frand0( int seed, float max )
{
  return ( float ) seed / ( float ) RAND_MAX * max ;
}

// return a random float value evenly distributed between min and max
__device__ float device_frand( int seed, float min, float max )
{
  float diff = max - min;
  return device_frand0( seed, diff ) + min;
}

// return an exponentially distributed random float
__device__ float device_erand( int seed, float inv_lambda )
{
  return -log( device_frand0( seed, 1.0 ) ) * inv_lambda;
}

// return a float value chosen from the normal distribution
// algorithm from polar method of G. E. P. Box, M. E. Muller, and G. Marsaglia, 
// as described by Donald E. Knuth in The Art of Computer Programming,
// Volume 2: Seminumerical Algorithms, section 3.4.1, subsection C, algorithm P
// also the implementation used by java.util.Random.nextGaussian
__device__ float device_grand0( int seed1, int seed2 )
{
  float v1, v2, s;
  
  do
  { 
    v1 = device_frand(seed1,-1,1);
    v2 = device_frand(seed2,-1,1);
    s = v1 * v1 + v2 * v2;
  }
  while ( s >= 1 || s == 0 );
  
  float multiplier = sqrtf(-2 * logf(s)/s);
  
  return v1 * multiplier;
}

// returns a random float value from a gaussian distribution
// with the given mean and sigma
__device__ float device_grand( int seed1, int seed2, float mean, float sigma )
{
  return mean + sigma * device_grand0( seed1, seed2 );
}

// calculates the probability density function for the gaussian
// distribution with given mean and sigma
__device__ float device_gvalue( float value, float mean, float sigma )
{
  float z = ( value - mean ) / sigma ;
  return expf( -0.5 * z * z ) / ( sqrtf( 2.0 * DEVICE_PI ) * sigma );
}
