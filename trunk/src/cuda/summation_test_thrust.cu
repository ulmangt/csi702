#include "hip/hip_runtime.h"
#include "filter_cuda_kernels.h"

#include <thrust/device_ptr.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/generate.h>
#include <thrust/reduce.h>
#include <thrust/functional.h>
#include <thrust/scan.h>
#include <cstdlib>

int small_rand( )
{
  return rand() % 10;
}

extern "C" void run_thrust_tests( float *data, int N, int reps )
{
  // alloate host array
  thrust::host_vector<float> host_data( N );

  int j;
  for ( j = 0 ; j < N ; j++ )
  {
    host_data[j] = data[j];
  }

  // copy data to device
  thrust::device_vector<float> device_data = host_data;


  // perform summation on device (gpu)
  time_t start = clock( );
  float sum;
  for ( j = 0 ; j < reps ; j++ )
  {
    sum = thrust::reduce(device_data.begin(), device_data.end(), 0.0f, thrust::plus<float>());
  }
  time_t end = clock( );

  double diff = (float) (end - start) / CLOCKS_PER_SEC;
  printf( "Parallel Sum (thrust): %f Time (sec): %0.5f\n", sum, diff );



  // allocate a device and host vector for the results
  thrust::device_vector<float> device_scan_data( N );
  thrust::host_vector<float> host_scan_data( N );

  // perform cumulative sum
  start = clock( );
  for ( j = 0 ; j < reps ; j++ )
  {
    thrust::inclusive_scan(device_data.begin(), device_data.end(), device_scan_data.begin());
  }
  end = clock( );

  // transfer partial sums from device (not included in timing test)
  host_scan_data = device_scan_data;

  diff = (float) (end - start) / CLOCKS_PER_SEC;
  printf( "Parallel Scan (thrust) Time (sec): %0.5f\n", diff );

  // don't print the cumulative sum array
  for ( j = 0 ; j < N ; j++ )
  {
    printf("%f\n", host_scan_data[j]);
  }
}
