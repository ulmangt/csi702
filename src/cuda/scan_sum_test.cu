#include "hip/hip_runtime.h"
#include "filter_cuda_kernels.h"

#include <thrust/device_ptr.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/generate.h>
#include <thrust/reduce.h>
#include <thrust/functional.h>
#include <thrust/scan.h>
#include <cstdlib>

int small_rand( )
{
  return rand() % 10;
}

int main( int argc, char** argv )
{
  int reps = 3000;
  int N = 1000000;

  // alloate host array
  thrust::host_vector<float> host_data( N );
  float data[N];

  // populate host array
  int i, j;
  for ( i = 0 ; i < N ; i++ )
  {
    data[i] = (float) rand() / (float) RAND_MAX;
  }

  for ( i = 0 ; i < N ; i++ )
  {
    host_data[i] = data[i];
  }

  // copy data to device
  thrust::device_vector<float> device_data = host_data;

  // perform summation on device (gpu)
  time_t start = clock( );
  float sum;
  for ( j = 0 ; j < reps ; j++ )
  {
    sum = thrust::reduce(device_data.begin(), device_data.end(), 0.0f, thrust::plus<float>());
  }
  time_t end = clock( );

  double diff = (float) (end - start) / CLOCKS_PER_SEC;
  printf( "Parallel Sum (thrust): %f Time (sec): %0.5f\n", sum, diff );

  // perform summation on host (cpu)
  start = clock( );
  float cpu_sum;
  for ( j = 0 ; j < reps ; j++ )
  {
    cpu_sum = 0.0f;
    for ( int i = 0 ; i < N ; i++ )
    {
      cpu_sum = cpu_sum + data[i];
    }
  }
  end = clock( );

  diff = (float) (end - start) / CLOCKS_PER_SEC;
  printf( "Serial Sum: %f Time (sec): %0.5f\n", cpu_sum, diff );

  // allocate a device and host vector for the results
  thrust::device_vector<float> device_scan_data( N );
  thrust::host_vector<float> host_scan_data( N );

  // perform cumulative sum
  start = clock( );
  for ( j = 0 ; j < reps ; j++ )
  {
    thrust::inclusive_scan(device_data.begin(), device_data.end(), device_scan_data.begin());
  }
  end = clock( );

  // transfer partial sums from device (not included in timing test)
  host_scan_data = device_scan_data;

  diff = (float) (end - start) / CLOCKS_PER_SEC;
  printf( "Parallel Scan (thrust) Time (sec): %0.5f\n", diff );

  // don't print the cumulative sum array 
  /*
  for ( i = 0 ; i < N ; i++ )
  {
    printf("%f\n", host_scan_data[i]);
  }
  */

  return 0;
}
