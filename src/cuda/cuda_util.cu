
#include <assert.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>

#include "cuda_constants.h"

extern "C" void copy_particles_host_to_device( int num )
{
  int size = sizeof( float ) * num;

  hipMemcpy( h_x_pos, d_x_pos, size, hipMemcpyHostToDevice );
  hipMemcpy( h_y_pos, d_y_pos, size, hipMemcpyHostToDevice );
  hipMemcpy( h_x_vel, d_x_vel, size, hipMemcpyHostToDevice );
  hipMemcpy( h_y_vel, d_y_vel, size, hipMemcpyHostToDevice );
  hipMemcpy( h_weight, d_weight, size, hipMemcpyHostToDevice );
  hipMemcpy( h_seed, d_seed, size, hipMemcpyHostToDevice );
}

extern "C" void copy_particles_device_to_host( int num )
{
  int size = sizeof( float ) * num;

  hipMemcpy( h_x_pos, d_x_pos, size, hipMemcpyDeviceToHost );
  hipMemcpy( h_y_pos, d_y_pos, size, hipMemcpyDeviceToHost );
  hipMemcpy( h_x_vel, d_x_vel, size, hipMemcpyDeviceToHost );
  hipMemcpy( h_y_vel, d_y_vel, size, hipMemcpyDeviceToHost );
  hipMemcpy( h_weight, d_weight, size, hipMemcpyDeviceToHost );
  hipMemcpy( h_seed, d_seed, size, hipMemcpyDeviceToHost );
}

// allocate memory for num particles on device
extern "C" void d_init_particle_mem( int num )
{
  int size = sizeof( float ) * num;

  hipMalloc( (void **) &d_x_pos, size );
  hipMalloc( (void **) &d_y_pos, size );
  hipMalloc( (void **) &d_x_vel, size );
  hipMalloc( (void **) &d_y_vel, size );
  hipMalloc( (void **) &d_weight, size );
  hipMalloc( (void **) &d_seed, size );
}

// free particle memory on device
extern "C" void d_free_particle_mem( )
{
  hipFree( d_x_pos );
  hipFree( d_y_pos );
  hipFree( d_x_vel );
  hipFree( d_y_vel );
  hipFree( d_weight );
  hipFree( d_seed );
}

// function adapted from CUDA Technical Training
extern "C" void checkCUDAError(const char *msg)
{
    hipError_t err = hipGetLastError();
    if( hipSuccess != err) 
    {
        fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString( err) );
        exit(EXIT_FAILURE);
    }                         
}
